#include "hip/hip_runtime.h"
/***********************************************************************[occurrence.cu]
Copyright(c) 2020, Muhammad Osama - Anton Wijs,
Copyright(c) 2022-present, Muhammad Osama.

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <https://www.gnu.org/licenses/>.
**********************************************************************************/

#include "solver.hpp"
#include "options.cuh"
#include "timer.cuh"
#include "grid.cuh"

namespace ParaFROST {

	__global__ void reduce_ot(const CNF* __restrict__ cnfptr, OT* __restrict__ ot)
	{
		for_parallel_x (tid, ot->size()) {
			OL& ol = (*ot)[tid];
			if (ol.size()) {
				const CNF& cnf = *cnfptr;
				S_REF* j = ol;
				forall_occurs(ol, i) {
					const S_REF ref = *i;
					if (!cnf[ref].deleted())
						*j++ = ref;
				}
				ol.resize(j - ol);
			}
		}
	}

	__global__ void reset_ot_k(OT* ot)
	{
		for_parallel_x (tid, ot->size()) {
			(*ot)[tid].clear();
		}
	}

	__global__ void create_ot_k(CNF* __restrict__ cnf, OT* __restrict__ ot_ptr)
	{
		for_parallel_x (tid, cnf->size()) {
			const S_REF r = cnf->ref(tid);
			SCLAUSE& c = (*cnf)[r];
			if (c.original() || c.learnt()) {
				OT& ot = *ot_ptr;
				forall_clause(c, lit) {
					ot[*lit].insert(r);
				}
			}
		}
	}

	void reduceOTAsync(CNF* cnf, OT* ot, const bool& print)
	{
		assert(cnf);
		assert(ot);
		if (gopts.profile_gpu) cutimer->start();
		OPTIMIZEBLOCKS(inf.nDualVars, BLOCK1D);
		reduce_ot << <nBlocks, BLOCK1D >> > (cnf, ot);
		if (print || gopts.sync_always) {
			LASTERR("Occurrence table reduction failed");
			SYNCALL;
			if (print) {
				LOGRULER('=', 30);
				LOG0("\toccurrence table");
				ot->print();
				LOGRULER('=', 30);
			}
		}
		if (gopts.profile_gpu) cutimer->stop(), cutimer->rot += cutimer->gpuTime();
	}

	void resetOTAsync(CNF* cnf, OT* ot)
	{
		assert(cnf);
		assert(ot);
		OPTIMIZEBLOCKS(inf.nDualVars, BLOCK1D);
		reset_ot_k << <nBlocks, BLOCK1D >> > (ot);
		if (gopts.sync_always) {
			LASTERR("Occurrence table reset failed");
			SYNCALL;
			assert(ot->accViolation(inf.maxVar));
		}
	}

	void createOTAsync(CNF* cnf, OT* ot, const bool& print)
	{
		assert(cnf);
		assert(ot);
		LOGN2(2, " Creating occurrence table on GPU..");
		if (gopts.profile_gpu) cutimer->start();
		resetOTAsync(cnf, ot);
		OPTIMIZEBLOCKS(inf.nClauses, BLOCK1D);
		create_ot_k << <nBlocks, BLOCK1D >> > (cnf, ot);
		if (print || gopts.sync_always) {
			LOG2(2, "");
			LASTERR("Occurrence table creation failed");
			SYNCALL;
			assert(ot->accViolation(inf.maxVar));
			if (print) {
				LOGRULER('=', 30);
				LOG0("\toccurrence table");
				ot->print();
				LOGRULER('=', 30);
			}
		}
		LOGDONE(2, 5);
		if (gopts.profile_gpu) cutimer->stop(), cutimer->cot += cutimer->gpuTime();
	}


	bool Solver::reallocOT(const hipStream_t& stream)
	{
		assert(inf.nLiterals);
		if (!flattenCNF(inf.nLiterals)) { simpstate = OTALLOC_FAIL; return false; }
		histSimp(inf.nLiterals);
		if (!cumm.resizeOTAsync(ot, inf.nLiterals, stream)) { simpstate = OTALLOC_FAIL; return false; }
		return true;
	}

}