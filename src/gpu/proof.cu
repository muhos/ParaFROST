#include "hip/hip_runtime.h"
/***********************************************************************[proof.cu]
Copyright(c) 2021, Muhammad Osama - Anton Wijs,
Copyright(c) 2022-present, Muhammad Osama.

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <https://www.gnu.org/licenses/>.
**********************************************************************************/

#include "count.cuh"
#include "proof.cuh"
#include "timer.cuh"
#include "shared.cuh"
#include "reduce.cuh"
#include "options.cuh"
#include "primitives.cuh"

using namespace ParaFROST;

//=======================================================
// constant proof lookup table
#define MAXLEADINGZEROS 30
__constant__
Byte BLUT[MAXLEADINGZEROS + 1] =
{
	1, 1, 1, 1, 1, 1,
	2, 2, 2, 2, 2, 2, 2,
	3, 3, 3, 3, 3, 3, 3,
	4, 4, 4, 4, 4, 4, 4,
	5, 5, 5, 5
};
#define COUNTBYTES(LIT) BLUT[MAXLEADINGZEROS - __clz(LIT)]
//=======================================================

__global__ void printHead(cuVecB* proof)
{
	printf("this = %p, mem = %p, size = %d, cap = %d", 
		proof, proof->data(), proof->size(), proof->capacity());
}

_PFROST_D_ void countBytes(const uint32& lit, Byte& perLit, uint32& total)
{
	// here we rely on data racing to avoid
	// counting the bytes for a duplicate
	// assuming perLit is initially 0
	if (!perLit) {
		Byte local;
		ORIGINIZELIT(orgLit, lit);
		perLit = local = COUNTBYTES(orgLit);
		total += local;
	}
	else // the more threads taking this path the better
		total += perLit;
	assert(total);
	assert(perLit >= 1 && perLit <= 5);
}

__global__ 
void cnt_proof(const uint32* __restrict__ literals, const uint32 numLits)
{
	uint32* sh_bytes = SharedMemory<uint32>();
	grid_t tid = global_tx_off;
	uint32 nbytes = 0;
	while (tid < numLits) {
		addr_t lbyte = DC_PTRS->d_lbyte;
		uint32 lit = literals[tid];
		countBytes(lit, lbyte[lit], nbytes);
		grid_t off = tid + blockDim.x;
		if (off < numLits) {
			lit = literals[off];
			countBytes(lit, lbyte[lit], nbytes);
		}
		tid += stride_x_off;
	}
	loadShared(sh_bytes, nbytes, numLits);
	sharedReduce(sh_bytes, nbytes);
	warpReduce(sh_bytes, nbytes);
	if (!threadIdx.x) devLBlocks[blockIdx.x] = nbytes;
}

__global__ 
void cnt_proof_verify(const uint32* __restrict__ literals, const uint32 numLits)
{
	grid_t tid = 0;
	while (tid < numLits) {
		addr_t lbyte = DC_PTRS->d_lbyte;
		const uint32 lit = literals[tid];
		if (lit & 0xF0000000) lbyte[lit] = 5;
		else if (lit & 0x0FE00000) lbyte[lit] = 4;
		else if (lit & 0x001FC000) lbyte[lit] = 3;
		else if (lit & 0x00003F80) lbyte[lit] = 2;
		else lbyte[lit] = 1;
		printf(" literal(%d) has %d bytes of its original\n", SIGN(lit) ? -int(ABS(lit)) : ABS(lit), lbyte[lit]);
		gcounter += lbyte[lit];
		tid++;
	}
	printf(" total = %d\n", gcounter);
}

uint32 cuPROOF::count(const uint32* literals, const uint32& numLits)
{
	if (!proof.checkFile()) LOGERROR("host proof system is not activated");
	if (!literals) return 0;
	if (!numLits) return 0;
	enabled = true;
	LOGN2(2, " Counting proof bytes..");
	OPTIMIZEBLOCKS2(numLits, BLOCK1D);
	OPTIMIZESHARED(blockSize, sizeof(uint32));
	SYNCALL; // sync any pending kernels or transfers
	if (gopts.profile_gpu) cutimer->start();
	cnt_proof << <nBlocks, blockSize, smemSize >> > (literals, numLits);
	LASTERR("Proof counting failed");
	CHECK(hipMemcpyFromSymbol(hostLBlocks, HIP_SYMBOL(devLBlocks), nBlocks * sizeof(uint32)));
	if (gopts.profile_gpu) cutimer->stop(), cutimer->ve += cutimer->gpuTime();
	const uint32 maxcap = seqreduceBlocks(hostLBlocks, nBlocks);
	assert(maxcap && maxcap < (numLits * sizeof(uint32)));
	LOGENDING(2, 5, "(%d bytes)", maxcap);
	return maxcap;
}

void cuPROOF::writeClause(addr_t& byte)
{
	assert(enabled);
	const size_t bytes = 16;
	const char delimiter = '0';
	char line[bytes];
	char* tail = line + bytes;
	*--tail = 0;
	while (*byte) {
		assert(byte != hostStream->end());
		uint32 ulit = 0, shift = 0;
		Byte b;
		do {
			b = *byte++;
			ulit |= (b & BYTEMASK) << shift;
			shift += 7;
		} while (b & BYTEMAX);
		const LIT_ST sign = SIGN(ulit);
		if (sign) proof.write('-');
		char* nstr = tail;
		assert(!*nstr);
		int digit = ABS(ulit);
		while (digit) {
			*--nstr = (digit % 10) + delimiter;
			digit /= 10;
		}
		while (nstr != tail) proof.write(*nstr++);
		proof.write(' ');
	}
	proof.write(delimiter);
	proof.write('\n');
}

void cuPROOF::writeProof(const hipStream_t& _s)
{
	if (!enabled) return;
	if (hostStream->empty()) return;
	SYNC(_s);
	assert(**hostStream == PROOF_ADDED || **hostStream == PROOF_DELETED);
	assert(!hostStream->back());
	size_t prevlines = deviceAdded;
	LOGN2(2, " Writing GPU proof data..");
	Byte* byte = *hostStream;
	Byte* end = hostStream->end();
	if (proof.isNonBinary()) {
		while (byte != end) {
			assert(*byte == PROOF_ADDED || *byte == PROOF_DELETED);
			if (*byte++ == PROOF_DELETED) {
				proof.write('d');
				proof.write(' ');
			}
			writeClause(byte);
			deviceAdded++;
			byte++; // skip 0
		}
	}
	else {
		while (byte != end) {
			const Byte b = *byte++;
			proof.write(b);
			if (!b) deviceAdded++;
		}
	}
	size_t lines = deviceAdded - prevlines;
	LOGENDING(2, 5, "(%zd clauses, %d bytes)", lines, hostStream->size());
	bytesWritten += hostStream->size();
	hostStream->clear();
}

bool cuPROOF::alloc(const uint32& maxcap)
{
	if (!maxcap) return false;
	assert(enabled);
	assert(hostPool.cap == devicePool.cap);
	const size_t header_size = sizeof(cuVecB);
	const size_t proof_cap   = maxcap;
	const size_t min_cap     = header_size + proof_cap;
	assert(min_cap);
	if (devicePool.cap < min_cap) {
		// device memory
		cumm.DFREE(devicePool);
		assert(devicePool.mem == NULL);
		if (!cumm.hasDeviceMem(min_cap, "Proof")) return false;
		CHECK(hipMalloc((void**)&devicePool.mem, min_cap));
		addr_t ea = devicePool.mem;
		deviceStream = (cuVecB*)ea, ea += header_size;
		header.alloc(ea, uint32(proof_cap)), ea += proof_cap;
		CHECK(hipMemcpyAsync(deviceStream, &header, header_size, hipMemcpyHostToDevice));
		assert(ea == devicePool.mem + min_cap);
		devicePool.cap = min_cap;
		// pinned host memory
		if (hostPool.mem) {
			CHECK(hipHostFree(hostPool.mem));
			hostPool.mem = NULL;
		}
		CHECK(hipHostMalloc((void**)&hostPool.mem, min_cap));
		ea = hostPool.mem;
		hostStream = (cuVecB*)ea, ea += header_size;
		hostStream->alloc(ea, uint32(proof_cap)), ea += proof_cap;
		assert(ea == hostPool.mem + min_cap);
		hostPool.cap = min_cap;
		SYNC(0); 
		header.clear(true);
	}
	return true;
}

void cuPROOF::cacheProof(const hipStream_t& _s) 
{
	if (!enabled) return;
	assert(hostStream);
	assert(deviceStream);
	CHECK(hipMemcpy(&header, deviceStream, sizeof(cuVecB), hipMemcpyDeviceToHost)); 
	const uint32 devSize = header.size();
	if (!devSize) return;
	header.clear();
	CHECK(hipMemcpyAsync(deviceStream, &header, sizeof(cuVecB), hipMemcpyHostToDevice, _s)); // reset gpu proof size
	assert(header.data());
	assert(hostStream->capacity() == header.capacity());
	hostStream->resize(devSize);
	if (gopts.profile_gpu) cutimer->start(_s);
	CHECK(hipMemcpyAsync(hostStream->data(), header.data(), devSize, hipMemcpyDeviceToHost, _s));
	if (gopts.sync_always) SYNC(_s);
	if (gopts.profile_gpu) cutimer->stop(_s), cutimer->ve += cutimer->gpuTime();
}

void cuPROOF::destroy()
{
	LOGN2(2, " Freeing up proof host-device memory..");
	cumm.DFREE(devicePool);
	if (hostPool.mem) {
		CHECK(hipHostFree(hostPool.mem));
		hostPool.mem = NULL, hostPool.cap = 0;
	}
	LOGDONE(2, 5);
}